
#include"mat_cuda.cuh"
#include<stdio.h>

hipblasHandle_t g_handle=NULL;

void matrixMul(double * x,int rowsX,int colsX,double*y,int rowsY,int colsY,double*z,int colsZ)
{
	hipblasStatus_t ret=HIPBLAS_STATUS_SUCCESS;
	if(g_handle==NULL)
		ret = hipblasCreate(&g_handle);
	if(ret != HIPBLAS_STATUS_SUCCESS)
		printf( "hipblasSgemm returned error code");

 	hipblasStatus_t stat; 
 	double alpha = 1.0; 
 	double beta = 0.0; 
 	stat = hipblasDgemm( 
 		g_handle,  
 		HIPBLAS_OP_N, 
 		HIPBLAS_OP_N, 
 		colsY, 
		rowsX, 
 		rowsY, 
 		&alpha, 
 		y, 
 		colsY, 
 		x, 
 		colsX, 
 		&beta, 
 		z, 
 		colsZ); 
 	hipDeviceSynchronize(); 
 	if(stat != HIPBLAS_STATUS_SUCCESS) 
	{ 
 		printf("matrixMulTA hipblasSgemm error\n"); 
		exit(0); 
	} 
}