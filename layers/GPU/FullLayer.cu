#include "hip/hip_runtime.h"
#include"FullLayer.cuh"

__device__ double fActiveFun(double src,int type)
{
	if(type==NL_RELU)
	{
		if(src<0.0)
			return 0.0;
		else
			return src;
	}
	else if(type==NL_SOFT_PLUS)
		return ::log(1+::exp(src));
	return 1/(1+::exp(-src));
}

__device__ double d_fActiveFun(double src,int type)
{
	if(type==NL_RELU)
	{
		if(src>0.0)
			return 1.0;
		else
			return 0.0;

	}
	else if(type==NL_SOFT_PLUS)
		return 1/(1+::exp(-src));
	return src*(1-src);
}

int CFullLayerGPU::initMem()
{
	m_weightLen=m_curNumFeature*m_inputNumFeature;

	DL_ASSER(m_weightLen!=0);

	hipError_t cudaStat;

	cudaStat=hipMalloc((void**)&m_delta,sizeof(double)*m_curNumFeature*batch);
	CUDA_ERROR(cudaStat);

	cudaStat=hipMalloc((void**)&m_afterDropWeight,sizeof(double)*m_inputNumFeature*m_curNumFeature);
	CUDA_ERROR(cudaStat);

	cudaStat=hipMalloc((void**)&m_fullData,sizeof(double)*m_curNumFeature*batch);
	CUDA_ERROR(cudaStat);

	cudaStat=hipMalloc((void**)&m_fullNoActiveData,sizeof(double)*m_curNumFeature*batch);
	CUDA_ERROR(cudaStat);

	cudaStat=hipMalloc((void**)&m_dropProbability,sizeof(float)*m_inputNumFeature*m_curNumFeature);
	CUDA_ERROR(cudaStat);

	hiprandStatus_t status=hiprandCreateGenerator(&m_hGen, HIPRAND_RNG_PSEUDO_DEFAULT);
	CURAND_ERROR(status);

	status=hiprandSetPseudoRandomGeneratorSeed(m_hGen,time(NULL));
	CURAND_ERROR(status);

	return CLayerGPU::initMem();
}

void CFullLayerGPU::freeMem()
{
	GPU_FREE(m_fullData);
	GPU_FREE(m_fullNoActiveData);
	GPU_FREE(m_dropProbability);
	GPU_FREE(m_afterDropWeight);

	hiprandDestroyGenerator(m_hGen);

	CLayerGPU::freeMem();
}

//block<<<weightLeng/threadNum>>>
//thread<<<min(1024,weightLeng)>>>
__global__ void dropOperator(float* dropRate,int leng,double rate)
{
	for(int i = 0; i < leng; i += blockDim.x * gridDim.x) 
	{ 
		int id = i + blockIdx.x * blockDim.x + threadIdx.x; 
		if(id < leng) 
		{
			if(rate==0)
				dropRate[id]=1;
			else if(dropRate[id]>rate)
				dropRate[id]=1;
			else
				dropRate[id]=0;
		}
	}
}

//block<<<weightLeng/threadNum>>>
//thread<<<min(1024,weightLeng)>>>
__global__ void weightOperator(double* afterWeight,double* weight,float* dropRate,int wLeng,double rate,bool bPre)
{
	for(int i = 0; i < wLeng; i += blockDim.x * gridDim.x) 
	{ 
		int id = i + blockIdx.x * blockDim.x + threadIdx.x; 
		if(id < wLeng) 
		{
			if(bPre)
				afterWeight[id]=weight[id]*(1-rate);
			else
				afterWeight[id]=weight[id]*dropRate[id];
		}
	}
}


//block<<<batch>>>
//thread<<<min(1024,numFeature)>>>
__global__ void feedForwardActive(double* activeData,double*notActiveData,double* bias,int numFeature,int activeType)
{
	int index=blockIdx.x*numFeature;
	for(int id = 0; id < numFeature; id += blockDim.x) 
	{ 
		int idx = id + threadIdx.x; 
		if(idx < numFeature)
			activeData[index+idx]=fActiveFun(notActiveData[index+idx]+bias[idx],activeType);
	}
}

void CFullLayerGPU::feedforward(double* srcData,DLparam& params)
{
	int activeType=params.activeType;
	bool bPred=params.pred;

	CURAND_ERROR(hiprandGenerateUniform(m_hGen,m_dropProbability,m_curNumFeature*m_inputNumFeature));

	hipError_t cudaStat=hipSuccess;
	int wLen=m_curNumFeature*m_inputNumFeature;
	dim3 threads = min(1024, wLen); 
	dim3 blocks  = min(1024, (wLen + threads.x - 1) / threads.x); 

	dropOperator<<<blocks,threads>>>(m_dropProbability,wLen,m_dropRate);
	cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);

	weightOperator<<<blocks,threads>>>(m_afterDropWeight,m_weight,m_dropProbability,wLen,m_dropRate,bPred);
	cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);

	matrixMulTA(srcData,batch,m_inputNumFeature,m_afterDropWeight,m_curNumFeature,m_inputNumFeature,m_fullNoActiveData,m_curNumFeature);

	blocks=batch;
	threads=min(1024,m_curNumFeature);
	feedForwardActive<<<blocks,threads>>>(m_fullData,m_fullNoActiveData,m_bias,m_curNumFeature,activeType);
	cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);
}


double CFullLayerGPU::getCost(DLparam& params)
{
	int dataSize=m_curNumFeature*m_inputNumFeature;
	double finSum=getWeightCost(m_weight,dataSize);
	return finSum*m_lambda/2;
}



//blocks<<<batch,NumFeature>>>
//thread<<<inputNumFeature>>>
__global__ void dFullActive(
						  double* deltaData,
						  double* deltaDataD,
						  double* fullNoActiveData,
						  int type
						  )
{
	int srcNo=blockIdx.x;
	int featureNo=threadIdx.x;
	int index=srcNo*blockDim.x+featureNo;
	deltaDataD[index]=deltaData[index]*d_fActiveFun(fullNoActiveData[index],type);
}

void CFullLayerGPU::backpropagation(double* preDeltas,DLparam& params)
{
	int activeType=params.activeType;
	dFullActive<<<batch,m_curNumFeature>>>(m_delta,
											  m_delta,
											  m_fullNoActiveData,
										      activeType);
	hipError_t cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);

	//��ΪCUDA����Ϊ��������������Ѵ����x,y��������֤�������Ϊ��
	matrixMul(m_delta,batch,m_curNumFeature,m_afterDropWeight,m_curNumFeature,m_inputNumFeature,preDeltas,m_inputNumFeature);
}

void CFullLayerGPU::getGrad(double* srcData)
{
	hipError_t cudaStat;
	matrixMulTB(m_delta,m_curNumFeature,srcData,batch,m_inputNumFeature,m_weightGrad,m_inputNumFeature);

	dim3 blocks(m_curNumFeature,m_inputNumFeature);

	int threadNum=min(MAX_THREAD_NUM,batch);

	int wLen=m_curNumFeature*m_inputNumFeature;
	dim3 threads2 = min(1024, wLen); 
	dim3 blocks2  = min(1024, (wLen + threads2.x - 1) / threads2.x); 

	fullWeightGrad2<<<blocks2,threads2>>>(m_weightGrad,m_weight,m_dropProbability,wLen,m_lambda,batch);
	cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);

	fullBiasGrad<<<m_curNumFeature,threadNum,sizeof(double)*threadNum>>>(m_delta,
							   m_biasGrad,
							   batch);
	cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);
}

void CFullLayerGPU::updateWeight(float mom,float alpha)
{
	int threadNum=min(1024,m_inputNumFeature);
	g_weightAndBiasAdd<<<m_curNumFeature,threadNum>>>(m_weight,m_weightGrad,
												      m_vecWeight,m_bias,
												      m_biasGrad,m_vecBias,
												      m_inputNumFeature,
												      mom,alpha);
	hipError_t cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);
}