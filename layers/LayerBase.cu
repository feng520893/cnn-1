#include "hip/hip_runtime.h"
#include"LayerBase.cuh"
#include<hipblas.h>

hipblasHandle_t g_handle=NULL;

__global__ void setGPUWeightValue(int index,double* weight,double value)
{
	weight[index]=value;
}

__global__ void getGPUWeightValue(double*value,int index,double* weight)
{
	*value=weight[index];
}

int CLayer::save(FILE* fp)
{
	hipError_t cudaStat;
	double* pTmp=new double[m_weightLen+m_curNumFeature];
	cudaStat=hipMemcpy(pTmp,m_weight,sizeof(double)*m_weightLen,hipMemcpyDeviceToHost);
	if(cudaStat!=hipSuccess)
	{	
		printf ("(Save file)device memory hipMemcpy failed\n"); 
		return -2;
	}
	cudaStat=hipMemcpy(pTmp+m_weightLen,m_bias,sizeof(double)*m_curNumFeature,hipMemcpyDeviceToHost);
	if(cudaStat!=hipSuccess)
	{	
		printf ("(Save file)device memory hipMemcpy failed\n"); 
		return -2;
	}
	fwrite(pTmp,sizeof(double)*(m_weightLen+m_curNumFeature),1,fp);
	delete [] pTmp;
	return PK_SUCCESS;
}

CLayer::CLayer()
{
	m_vecBias=m_vecWeight=m_weight=m_bias=m_weightGrad=m_biasGrad=m_delta=NULL;
}

int CLayer::load(FILE*fp)
{
	hipError_t cudaStat;
	double* pTmp=new double[m_weightLen+m_curNumFeature];
	fread(pTmp,sizeof(double)*(m_weightLen+m_curNumFeature),1,fp);
	cudaStat=hipMemcpy(m_weight,pTmp,sizeof(double)*m_weightLen,hipMemcpyHostToDevice);
	if(cudaStat!=hipSuccess)
	{	
		printf ("(Load file)device memory hipMemcpy failed\n"); 
		return -2;
	}
	cudaStat=hipMemcpy(m_bias,pTmp+m_weightLen,sizeof(double)*m_curNumFeature,hipMemcpyHostToDevice);
	if(cudaStat!=hipSuccess)
	{	
		printf ("(Load file)device memory hipMemcpy failed\n"); 
		return -2;
	}
	delete [] pTmp;
	return PK_SUCCESS;
}

int CLayer::initMem()
{
	hipError_t cudaStat;
	cudaStat=hipMalloc((void**)&m_vecBias,sizeof(double)*m_curNumFeature);
	if(cudaStat!=hipSuccess)
	{
		printf ("device memory hipMalloc failed\n"); 
		freeMem();
		return -1;
	}
	cudaStat=hipMemset(m_vecBias,0,sizeof(double)*m_curNumFeature);
	if(cudaStat!=hipSuccess)
	{
		printf ("device memory hipMemset failed\n"); 
		freeMem();
		return -2;
	}

	cudaStat=hipMalloc((void**)&m_bias,sizeof(double)*m_curNumFeature);
	if(cudaStat!=hipSuccess)
	{
		printf ("device memory hipMalloc failed\n"); 
		freeMem();
		return -1;
	}
	cudaStat=hipMemset(m_bias,0,sizeof(double)*m_curNumFeature);
	if(cudaStat!=hipSuccess)
	{
		printf ("device memory hipMemset failed\n"); 
		freeMem();
		return -2;
	}

	cudaStat=hipMalloc((void**)&m_biasGrad,sizeof(double)*m_curNumFeature);
	if(cudaStat!=hipSuccess)
	{
		printf ("device memory hipMalloc failed\n"); 
		freeMem();
		return -1;
	}

	cudaStat=hipMalloc((void**)&m_weight,sizeof(double)*m_weightLen);
	if(cudaStat!=hipSuccess)
	{
		printf ("device memory hipMalloc failed\n");  
		freeMem();
		return -1;
	}
	double* pData=new double[m_weightLen];
	randn(pData,m_weightLen);
	cudaStat=hipMemcpy(m_weight,pData,sizeof(double)*m_weightLen, hipMemcpyHostToDevice);
	delete [] pData;
	if(cudaStat!=hipSuccess)
	{
		printf ("device memory hipMemcpy failed\n"); 
		freeMem();
		return -1;
	}

	cudaStat=hipMalloc((void**)&m_weightGrad,sizeof(double)*m_weightLen);
	if(cudaStat!=hipSuccess)
	{
		printf ("device memory hipMalloc failed\n");  
		freeMem();
		return -1;
	}

	cudaStat=hipMalloc((void**)&m_vecWeight,sizeof(double)*m_weightLen);
	if(cudaStat!=hipSuccess)
	{
		printf("device memory hipMalloc failed\n");  
		freeMem();
		return -1;
	}

	cudaStat=hipMemset(m_vecWeight,0,sizeof(double)*m_weightLen);
	if(cudaStat!=hipSuccess)
	{
		printf("device memory hipMemset failed\n");  
		freeMem();
		return -2;
	}
	return PK_SUCCESS;
}

void CLayer::freeMem()
{
	GPU_FREE(m_vecBias);
	GPU_FREE(m_vecWeight);
	GPU_FREE(m_weight);
	GPU_FREE(m_bias);
	GPU_FREE(m_weightGrad);
	GPU_FREE(m_biasGrad);
	GPU_FREE(m_delta);
}

void CLayer::setWeightValue(int index,double value)
{
	setGPUWeightValue<<<1,1>>>(index,m_weight,value);
}

double CLayer::getWeightValue(int index)
{
	double*value=NULL;
	hipMalloc((void**)&value,sizeof(double));
	getGPUWeightValue<<<1,1>>>(value,index,m_weight);
	double result=0.0;
	hipMemcpy(&result,value,sizeof(double),hipMemcpyDeviceToHost);
	hipFree(value);
	return result;
}

__global__ void setGPUBiasValue(int index,double* bias,double value)
{
	bias[index]=value;
}

__global__ void getGPUBiasValue(double*value,int index,double* bias)
{
	*value=bias[index];
}


void CLayer::setBiasValue(int index,double value)
{
	setGPUBiasValue<<<1,1>>>(index,m_bias,value);
}

double CLayer::getBiasValue(int index)
{
	double*value=NULL;
	hipMalloc((void**)&value,sizeof(double));
	getGPUBiasValue<<<1,1>>>(value,index,m_bias);
	double result=0.0;
	hipMemcpy(&result,value,sizeof(double),hipMemcpyDeviceToHost);
	hipFree(value);
	return result;
}

//���μӷ�����Ȩֵ��ƽ����
__global__ void sumOfSquares(double *num,unsigned int dataSize, double* result)
{
    extern __shared__ double shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

	int THREAD_NUM=blockDim.x;
	int BLOCK_NUM=gridDim.x;

    shared[tid] = 0;
    for(int i = bid * THREAD_NUM + tid; i < dataSize;
        i += BLOCK_NUM * THREAD_NUM) {
       shared[tid] += num[i] * num[i];
    }

    __syncthreads();
    int offset = THREAD_NUM / 2;
    while(offset > 0) {
        if(tid < offset) {
            shared[tid] += shared[tid + offset];
        }
        offset >>= 1;
        __syncthreads();
    }

    if(tid == 0) {
        result[bid] = shared[0];
    }
}

double getWeightCost(double* devWeight,unsigned int dataSize)
{
	const int BLOCK_NUM=32;
	const int THREAD_NUM=256;

	double* result=NULL;

	hipMalloc((void**) &result, sizeof(double) * BLOCK_NUM);

	sumOfSquares<<<BLOCK_NUM, THREAD_NUM,
        THREAD_NUM * sizeof(double)>>>(devWeight,dataSize,result);

	hipDeviceSynchronize();

	double sum[BLOCK_NUM];
    hipMemcpy(sum, result, sizeof(double) * BLOCK_NUM,hipMemcpyDeviceToHost);

    hipFree(result);

   double finSum = 0;
    for(int i = 0; i < BLOCK_NUM; i++)
        finSum += sum[i];
	return finSum;
}

//block<<<weightLeng/threadNum>>>
//thread<<<min(1024,weightLeng)>>>
__global__ void fullWeightGrad2(double* wgrad, double* weight,float* dropW,int wLeng, double lambda, int batch) 
{ 
	for(int i = 0; i < wLeng; i += blockDim.x * gridDim.x) 
	{ 
		int id = i + blockIdx.x * blockDim.x + threadIdx.x; 
		if(id < wLeng) 
			wgrad[id] = (wgrad[id] / batch + lambda * weight[id])*dropW[id]; 
	}
}

//block<<<numFeature,inputNumFeature>>>
//thread<<<1>>>
__global__ void fullWeightGrad(double* wgrad, double* weight, double lambda, int batch) 
{ 
	int id=blockIdx.x*gridDim.y+blockIdx.y;
	wgrad[id] = wgrad[id] / batch + lambda * weight[id]; 
}

//block<<<numFeature>>>
//thread<<<min(1024,batch)>>>
//share<<<sizeof(double)*threadNum>>>
__global__ void fullBiasGrad( double* delta, double* grads,int batch)
{
	extern __shared__ double _sum[];
	int featureNo=blockIdx.x;
	int tid=threadIdx.x;

	_sum[tid]=0.0;

	for(int i=tid;i<batch;i+=blockDim.x)
	{
		int deltaIndex=i*gridDim.x+featureNo;
		_sum[tid]+=delta[deltaIndex];
	}

	int len = blockDim.x; 
	 while(len != 1) 
	 { 
		 __syncthreads(); 
		 int skip = (len + 1) >> 1; 
		 if(threadIdx.x < (len >> 1)) 
		 { 
			 _sum[threadIdx.x] +=_sum[threadIdx.x + skip]; 
		 } 
		 len = (len + 1) >> 1; 
	 } 

	 if(threadIdx.x==0)
		grads[featureNo]=_sum[0]/batch;
}

//block<<<curNumFeature>>>
//thread<<<min(maxThread,oneFeatureWeightSize)>>>
__global__ void g_weightAndBiasAdd(double* weights,
								 double* weightGrads,
								 double* vec_weight,
								 double* bias,
								 double* biasGrad,
								 double* vec_bias,
								 int oneFeatureWeightSize,
								 float mom,
								 float alpha)
{
	int featureNo=blockIdx.x;
	for(int i=threadIdx.x;i<oneFeatureWeightSize;i+=blockDim.x)
	{
		int index=oneFeatureWeightSize*featureNo+i;
		vec_weight[index]=vec_weight[index]*mom+weightGrads[index]*alpha;
		weights[index]=weights[index]-vec_weight[index];
	}
	if(threadIdx.x==0)
	{
		vec_bias[featureNo]=vec_bias[featureNo]*mom+biasGrad[featureNo]*alpha;
		bias[featureNo]-=vec_bias[featureNo];
	}
}


//block<<<batch,numFeature>>>
//thread<<<min(maxThreadNum,inputNumFeature)>>>
//share<<sizeof(double)*threadNum>>>
__global__ void g_fullConnect(
						   double* srcData,
						   double* weight,
						   int inputNumFeature,
						   double* fullData,
						   double* bias
						   )
{
	extern __shared__ double featureSum[];

	int srcNo=blockIdx.x;
	int featureNo=blockIdx.y;
	int tId=threadIdx.x;
	
	int fullDataIndex=srcNo*gridDim.y+featureNo;

	featureSum[tId]=0;
	__syncthreads();

	for(int i=tId;i<inputNumFeature;i+=blockDim.x)
	{
		int dataIndex=srcNo*inputNumFeature+i;
		int weightIndex=featureNo*inputNumFeature+i;
		featureSum[tId]+=srcData[dataIndex]*weight[weightIndex];
	}
	 __syncthreads();

	 int len = blockDim.x; 
	 while(len != 1) 
	 { 
		 __syncthreads(); 
		 int skip = (len + 1) >> 1; 
		 if(threadIdx.x < (len >> 1)) 
		 { 
			 featureSum[threadIdx.x] +=featureSum[threadIdx.x + skip]; 
		 } 
		 len = (len + 1) >> 1; 
	 } 
	 __syncthreads();

	 if(threadIdx.x==0)
		 fullData[fullDataIndex]=featureSum[0]+bias[featureNo];
}



void matrixMulTB(double * x,int colsX,double*y,int rowsY,int colsY,double*z,int colsZ) 
{  
	hipblasStatus_t ret=HIPBLAS_STATUS_SUCCESS;
	if(g_handle==NULL)
		ret = hipblasCreate(&g_handle);
	if(ret != HIPBLAS_STATUS_SUCCESS)
		printf( "hipblasSgemm returned error code");

 	hipblasStatus_t stat; 
 	double alpha = 1.0; 
 	double beta = 0.0; 
 	stat = hipblasDgemm( 
 		g_handle,  
 		HIPBLAS_OP_N, 
 		HIPBLAS_OP_T, 
 		colsY, 
		colsX, 
 		rowsY, 
 		&alpha, 
 		y, 
 		colsY, 
 		x, 
 		colsX, 
 		&beta, 
 		z, 
 		colsZ); 
 	hipDeviceSynchronize(); 
 	if(stat != HIPBLAS_STATUS_SUCCESS) 
	{ 
 		printf("matrixMulTA hipblasSgemm error\n"); 
		exit(0); 
	} 
} 

void matrixMul(double * x,int rowsX,int colsX,double*y,int rowsY,int colsY,double*z,int colsZ) 
{  
	hipblasStatus_t ret=HIPBLAS_STATUS_SUCCESS;
	if(g_handle==NULL)
		ret = hipblasCreate(&g_handle);
	if(ret != HIPBLAS_STATUS_SUCCESS)
		printf( "hipblasSgemm returned error code");

 	hipblasStatus_t stat; 
 	double alpha = 1.0; 
 	double beta = 0.0; 
 	stat = hipblasDgemm( 
 		g_handle,  
 		HIPBLAS_OP_N, 
 		HIPBLAS_OP_N, 
 		colsY, 
		rowsX, 
 		rowsY, 
 		&alpha, 
 		y, 
 		colsY, 
 		x, 
 		colsX, 
 		&beta, 
 		z, 
 		colsZ); 
 	hipDeviceSynchronize(); 
 	if(stat != HIPBLAS_STATUS_SUCCESS) 
	{ 
 		printf("matrixMulTA hipblasSgemm error\n"); 
		exit(0); 
	} 
} 

void matrixMulTA(double * x,int rowsX,int colsX,double*y,int rowsY,int colsY,double*z,int colsZ) 
{  
	hipblasStatus_t ret=HIPBLAS_STATUS_SUCCESS;
	if(g_handle==NULL)
		ret = hipblasCreate(&g_handle);
	if(ret != HIPBLAS_STATUS_SUCCESS)
		printf( "hipblasSgemm returned error code");

 	hipblasStatus_t stat; 
 	double alpha = 1.0; 
 	double beta = 0.0; 
 	stat = hipblasDgemm( 
		g_handle,  
 		HIPBLAS_OP_T, 
 		HIPBLAS_OP_N, 
 		rowsY, 
 		rowsX, 
 		colsY, 
 		&alpha, 
 		y, 
 		colsY, 
 		x, 
 		colsX, 
 		&beta, 
 		z, 
 		colsZ); 
 	hipDeviceSynchronize(); 
 	if(stat != HIPBLAS_STATUS_SUCCESS) 
	{ 
 		printf("matrixMulTA hipblasSgemm error\n"); 
		exit(0); 
	} 
} 