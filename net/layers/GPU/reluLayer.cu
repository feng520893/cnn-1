#include "hip/hip_runtime.h"
#include"relulayer.cuh"

//block<<<weightLeng/threadNum>>>
//thread<<<min(1024,Leng)>>>
__global__ void activeRelu(precision* src, precision* dest,unsigned int size) 
{ 
	for(int i = 0; i < size; i += blockDim.x * gridDim.x) 
	{ 
		int id = i + blockIdx.x * blockDim.x + threadIdx.x; 
		if(id < size) 
		{
			dest[id] = ::max(0.0,src[id]);
		}
	}
}


precision CReluLayerGPU::feedforward(std::vector<Blob<precision>*>& bottoms,std::vector<Blob<precision>*>& tops)
{
	int size=bottoms[0]->size();
	dim3 threads2 = min(1024, size); 
	dim3 blocks2  = min(65535,(size + threads2.x - 1) / threads2.x); 
	activeRelu<<<blocks2,threads2>>>(bottoms[0]->gpuData,tops[0]->gpuData,size);
	hipError_t cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);
	return 0;
}

//block<<<weightLeng/threadNum>>>
//thread<<<min(1024,Leng)>>>
__global__ void d_activeRelu(precision* src, precision* dest,unsigned int size) 
{ 
	for(int i = 0; i < size; i += blockDim.x * gridDim.x) 
	{ 
		int id = i + blockIdx.x * blockDim.x + threadIdx.x; 
		if(id < size) 
		{
			if(src[id]>0.0)
				dest[id] = 1.0;
			else
			    dest[id] = 0.0;
		}
	}
}

int CReluLayerGPU::backpropagation(std::vector<Blob<precision>*>& tops,std::vector<bool>& propagateDown,std::vector<Blob<precision>*>& bottoms)
{
	int size=bottoms[0]->size();
	dim3 threads2 = min(1024, size); 
	dim3 blocks2  = min(65535, (size + threads2.x - 1) / threads2.x); 
	d_activeRelu<<<blocks2,threads2>>>(tops[0]->gpuData,bottoms[0]->gpuData,size);
	hipError_t cudaStat=hipDeviceSynchronize();
	CUDA_ERROR(cudaStat);
	return NET_SUCCESS;
}